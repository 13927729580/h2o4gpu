#include "matrix/matrix.h"
#include "matrix/matrix_dense.h"
#include <thrust/device_vector.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include <cstdlib>
#include "h2oaikmeans.h"
#include "kmeans.h"

#define CUDACHECK(cmd) do {                         \
    hipError_t e = cmd;                              \
    if( e != hipSuccess ) {                          \
      printf("Cuda failure %s:%d '%s'\n",             \
             __FILE__,__LINE__,hipGetErrorString(e));   \
      exit(EXIT_FAILURE);                             \
    }                                                 \
  } while(0)


template<typename T>
void fill_array(T& array, int m, int n) {
  for(int i = 0; i < m; i++) {
    for(int j = 0; j < n; j++) {
      array[i * n + j] = (i % 2)*3 + j;
    }
  }
}

template<typename T>
void random_data(thrust::device_vector<T>& array, int m, int n) {
  thrust::host_vector<T> host_array(m*n);
  for(int i = 0; i < m * n; i++) {
    host_array[i] = (T)rand()/(T)RAND_MAX;
  }
  array = host_array;
}

void random_labels(thrust::device_vector<int>& labels, int n, int k) {
  thrust::host_vector<int> host_labels(n);
  for(int i = 0; i < n; i++) {
    host_labels[i] = rand() % k;
  }
  labels = host_labels;
}

#define __HBAR__ \
"----------------------------------------------------------------------------\n"

namespace h2oaikmeans {

    template <typename T>
    H2OAIKMeans<T>::H2OAIKMeans(const T* A, int k, size_t n, size_t d)
    {
      _A = A; _k = k; _n = n; _d = d;
    }

    template <typename T>
    int H2OAIKMeans<T>::Solve() {
      int max_iterations = 10000;
      int n = 260753;  // rows
      int d = 298;  // cols
      int k = 100;  // clusters
      double thresh = 1e-3;  // relative improvement

      int n_gpu;
      hipGetDeviceCount(&n_gpu);
      std::cout << n_gpu << " gpus." << std::endl;

      thrust::device_vector<T> *data[16];
      thrust::device_vector<int> *labels[16];
      thrust::device_vector<T> *centroids[16];
      thrust::device_vector<T> *distances[16];
      for (int q = 0; q < n_gpu; q++) {
        hipSetDevice(q);
        data[q] = new thrust::device_vector<T>(n/n_gpu*d);
        labels[q] = new thrust::device_vector<int>(n/n_gpu*d);
        centroids[q] = new thrust::device_vector<T>(k * d);
        distances[q] = new thrust::device_vector<T>(n);
      }

      std::cout << "Generating random data" << std::endl;
      std::cout << "Number of points: " << n << std::endl;
      std::cout << "Number of dimensions: " << d << std::endl;
      std::cout << "Number of clusters: " << k << std::endl;
      std::cout << "Max. number of iterations: " << max_iterations << std::endl;
      std::cout << "Stopping threshold: " << thresh << std::endl;

      for (int q = 0; q < n_gpu; q++) {
        random_data<T>(*data[q], n/n_gpu, d);
        random_labels(*labels[q], n/n_gpu, k);
      }

      double t0 = timer<double>();
      kmeans::kmeans<T>(n, d, k, data, labels, centroids, distances, n_gpu, max_iterations, true, thresh);
      double time = static_cast<double>(timer<double>() - t0);
      std::cout << "  Time: " << time << " s" << std::endl;

      for (int q = 0; q < n_gpu; q++) {
        delete(data[q]);
        delete(labels[q]);
        delete(centroids[q]);
        delete(distances[q]);
      }
      return 0;
    }

    template <typename T>
    int makePtr_dense(int n_gpu, size_t rows, size_t cols, const char ord, int k, int max_iterations, T threshold, const T* srcdata, const int* srclabels, void ** res) {
      int n=rows;
      int d=cols;

      thrust::device_vector<T> *data[n_gpu];
      thrust::device_vector<int> *labels[n_gpu];
      thrust::device_vector<T> *centroids[n_gpu];
      thrust::device_vector<T> *distances[n_gpu];
      for (int q = 0; q < n_gpu; q++) {
        CUDACHECK(hipSetDevice(q));
        data[q] = new thrust::device_vector<T>(n/n_gpu*d);
        labels[q] = new thrust::device_vector<int>(n/n_gpu*d);
        centroids[q] = new thrust::device_vector<T>(k * d);
        distances[q] = new thrust::device_vector<T>(n);
      }

      std::cout << "Number of points: " << n << std::endl;
      std::cout << "Number of dimensions: " << d << std::endl;
      std::cout << "Number of clusters: " << k << std::endl;
      std::cout << "Max. number of iterations: " << max_iterations << std::endl;
      std::cout << "Stopping threshold: " << threshold << std::endl;

      bool init_from_labels=true;
#if(1)
      for (int q = 0; q < n_gpu; q++) {
        CUDACHECK(hipSetDevice(q));
        std::cout << "Copying data to device: " << q << std::endl;
#if(0)
        CUDACHECK(hipMemcpy(data[q]->data().get(), &srcdata[q*n/n_gpu*d], sizeof(T)*n/n_gpu*d, hipMemcpyHostToDevice));
        std::cout << "Done copying data to device: " << q << " of bytes size " << sizeof(T)*n/n_gpu*d << std::endl;
        CUDACHECK(hipMemcpy(labels[q]->data().get(), &srclabels[q*n/n_gpu*d], sizeof(T)*n/n_gpu*d, hipMemcpyHostToDevice));
        std::cout << "Done copying labels to device: " << q << " of bytes size " << sizeof(T)*n/n_gpu*d << std::endl;
#else
        thrust::copy(&srcdata[q*n/n_gpu*d],&srcdata[(q+1)*n/n_gpu*d],data[q]->begin());
        //        thrust::copy(&srclabels[q*n/n_gpu],&srclabels[(q+1)*n/n_gpu],labels[q]->begin()); // why 
        random_labels(*labels[q], n/n_gpu, k); // why only portion of lables set?
#endif
      }
#else
      for (int q = 0; q < n_gpu; q++) {
        random_data<T>(*data[q], n/n_gpu, d);
        //        random_labels(*labels[q], n/n_gpu*d, k);
        random_labels(*labels[q], n/n_gpu, k); // why only portion of lables set?
      }
#endif

      double t0 = timer<double>();
      kmeans::kmeans<T>(n,d,k,data,labels,centroids,distances,n_gpu,max_iterations,init_from_labels,threshold);
      double time = static_cast<double>(timer<double>() - t0);
      std::cout << "  Time: " << time << " s" << std::endl;

      // copy result of centroids (sitting entirely on each device) back to host
      thrust::host_vector<T> *ctr = new thrust::host_vector<T>(*centroids[0]);
      //      hipMemcpy(ctr->data().get(), centroids[0]->data().get(), sizeof(T)*k*d, hipMemcpyDeviceToHost);
      *res = ctr->data();
      
      for (int q = 0; q < n_gpu; q++) {
        delete(data[q]);
        delete(labels[q]);
        //        delete(centroids[q]);
        //        delete(distances[q]);
      }

      return 0;
    }
    template int
    makePtr_dense<float>(int n_gpu, size_t rows, size_t cols, const char ord, int k, int max_iterations, float threshold, const float *srcdata, const int *srclabels, void **a);


// Explicit template instantiation.
//#if !defined(H2OAIGLM_DOUBLE) || H2OAIGLM_DOUBLE==1
//template class H2OAIKMeans<double>;
//#endif

#if !defined(H2OAIGLM_SINGLE) || H2OAIGLM_SINGLE==1
    template class H2OAIKMeans<float>;
#endif

}  // namespace h2oaikmeans

#ifdef __cplusplus
extern "C" {
#endif

  int make_ptr_float_kmeans(int n_gpu, size_t mTrain, size_t n, const char ord, int k, int max_iterations, float threshold, const float* srcdata, const int* srclabels, void** res) {
    return h2oaikmeans::makePtr_dense<float>(n_gpu, mTrain, n, ord, k, max_iterations, threshold, srcdata, srclabels, res);
}

#ifdef __cplusplus
}
#endif
