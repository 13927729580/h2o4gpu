#include "matrix/matrix.h"
#include "matrix/matrix_dense.h"
#include <thrust/device_vector.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include <cstdlib>
#include "h2oaikmeans.h"
#include "kmeans.h"

typedef float real_t;
template<typename T>
void fill_array(T& array, int m, int n) {
  for(int i = 0; i < m; i++) {
    for(int j = 0; j < n; j++) {
      array[i * n + j] = (i % 2)*3 + j;
    }
  }
}

template<typename T>
void random_data(thrust::device_vector<T>& array, int m, int n) {
  thrust::host_vector<T> host_array(m*n);
  for(int i = 0; i < m * n; i++) {
    host_array[i] = (T)rand()/(T)RAND_MAX;
  }
  array = host_array;
}

void random_labels(thrust::device_vector<int>& labels, int n, int k) {
  thrust::host_vector<int> host_labels(n);
  for(int i = 0; i < n; i++) {
    host_labels[i] = rand() % k;
  }
  labels = host_labels;
}

#define __HBAR__ \
"----------------------------------------------------------------------------\n"

namespace h2oaikmeans {

template <typename M>
H2OAIKMeans<M>::H2OAIKMeans(const M* A, int k, size_t n, size_t d)
{
_A = A; _k = k; _n = n; _d = d;
}

template <typename M>
int H2OAIKMeans<M>::Solve() {
  int max_iterations = 10000;
  int n = 260753;  // rows
  int d = 298;  // cols
  int k = 100;  // clusters
  double thresh = 1e-3;  // relative improvement

  int n_gpu;
  hipGetDeviceCount(&n_gpu);
  std::cout << n_gpu << " gpus." << std::endl;

  thrust::device_vector<real_t> *data[16];
  thrust::device_vector<int> *labels[16];
  thrust::device_vector<real_t> *centroids[16];
  thrust::device_vector<real_t> *distances[16];
  for (int q = 0; q < n_gpu; q++) {
    hipSetDevice(q);
    data[q] = new thrust::device_vector<real_t>(n/n_gpu*d);
    labels[q] = new thrust::device_vector<int>(n/n_gpu*d);
    centroids[q] = new thrust::device_vector<real_t>(k * d);
    distances[q] = new thrust::device_vector<real_t>(n);
  }

  std::cout << "Generating random data" << std::endl;
  std::cout << "Number of points: " << n << std::endl;
  std::cout << "Number of dimensions: " << d << std::endl;
  std::cout << "Number of clusters: " << k << std::endl;
  std::cout << "Max. number of iterations: " << max_iterations << std::endl;
  std::cout << "Stopping threshold: " << thresh << std::endl;

  for (int q = 0; q < n_gpu; q++) {
    random_data<real_t>(*data[q], n/n_gpu, d);
    random_labels(*labels[q], n/n_gpu, k);
  }

    double t0 = timer<double>();
    kmeans::kmeans<real_t>(n, d, k, data, labels, centroids, distances, n_gpu, max_iterations, true, thresh);
    double time = static_cast<double>(timer<double>() - t0);
    std::cout << "  Time: " << time << " s" << std::endl;

    for (int q = 0; q < n_gpu; q++) {
      delete(data[q]);
      delete(labels[q]);
      delete(centroids[q]);
      delete(distances[q]);
    }
    return 0;
  }

// Explicit template instantiation.
#if !defined(H2OAIGLM_DOUBLE) || H2OAIGLM_DOUBLE==1
template class H2OAIKMeans<double>;
#endif

#if !defined(H2OAIGLM_SINGLE) || H2OAIGLM_SINGLE==1
template class H2OAIKMeans<float>;
#endif

}  // namespace h2oaikmeans

