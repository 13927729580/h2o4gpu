#include "hip/hip_runtime.h"
#include <hipblas.h>

#include "cml/cml_blas.cuh"
#include "cml/cml_matrix.cuh"
#include "cml/cml_vector.cuh"
#include "equil_helper.cuh"
#include "matrix/matrix.h"
#include "matrix/matrix_dense.h"
#include "util.h"
#include "timer.h"

extern int checkwDev(int wDev);

namespace pogs {

////////////////////////////////////////////////////////////////////////////////
////////////////////////////// Helper Functions ////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
namespace {

// File scoped constants.
const NormTypes kNormEquilibrate = kNorm2; 
const NormTypes kNormNormalize   = kNormFro;

template<typename T>
struct GpuData {
  const T *orig_data; // pointer to data on CPU
  hipblasHandle_t handle; // handle for data on GPU
  GpuData(const T *orig_data) : orig_data(orig_data) {
    hipblasCreate(&handle);
    DEBUG_CUDA_CHECK_ERR();
  }
  ~GpuData() {
    if(handle!=NULL) hipblasDestroy(handle);
    DEBUG_CUDA_CHECK_ERR();
  }
};

hipblasOperation_t OpToCublasOp(char trans) {
  ASSERT(trans == 'n' || trans == 'N' || trans == 't' || trans == 'T');
  return trans == 'n' || trans == 'N' ? HIPBLAS_OP_N : HIPBLAS_OP_T;
}

template <typename T>
T NormEst(hipblasHandle_t hdl, NormTypes norm_type, const MatrixDense<T>& A);

template <typename T>
void MultDiag(const T *d, const T *e, size_t m, size_t n,
              typename MatrixDense<T>::Ord ord, T *data);

}  // namespace

////////////////////////////////////////////////////////////////////////////////
/////////////////////// MatrixDense Implementation /////////////////////////////
////////////////////////////////////////////////////////////////////////////////

  // original MatrixDense where only trainX and no trainY or validX or validY
  // Used by elastic_net.cpp to pass CPU data and put on GPU
template <typename T>
MatrixDense<T>::MatrixDense(int wDev, char ord, size_t m, size_t n, const T *data)
  : Matrix<T>(m, n, 0), _wDev(wDev), _datatype(0),_data(0) {
  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));
  _datay=NULL;
  _vdata=NULL;
  _vdatay=NULL;

  ASSERT(ord == 'r' || ord == 'R' || ord == 'c' || ord == 'C');
  _ord = (ord == 'r' || ord == 'R') ? ROW : COL;

  
  fprintf(stderr,"MatrixDense1: ord=%c m=%d n=%d\n",ord,(int)m,(int)n);
  
#ifdef _DEBUG
  //    CUDACHECK(hipSetDeviceFlags(hipDeviceMapHost)); // TODO: MapHostMemory
  hipDeviceProp_t props;
  CUDACHECK(hipGetDeviceProperties(&props, _wDev));
  fprintf(stderr,"Using: Compute %d.%d CUDA device: [%s] with id=%2d\n", props.major, props.minor, props.name,wDev); fflush(stderr);
#endif
  
  // Set GPU specific _info.


  PUSH_RANGE("MDnew",MDnew,1);
  GpuData<T> *info = new GpuData<T>(data); // new structure (holds pointer to data and GPU handle)
  this->_info = reinterpret_cast<void*>(info);
  POP_RANGE("MDnew",MDnew,1);

  // Copy Matrix to GPU.
  PUSH_RANGE("MDsend",MDsend,1);
  //  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info); // cast void -> GpuData
  double t0 = timer<double>();
  hipMalloc(&_data, this->_m * this->_n * sizeof(T)); // allocate on GPU
  double t1 = timer<double>();
  hipMemcpy(_data, info->orig_data, this->_m * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
  double t2 = timer<double>();
  printf("Time to allocate the data matrix on the GPU: %f\n", t1-t0);
  printf("Time to copy the data matrix to the GPU    : %f\n", t2-t1);
  POP_RANGE("MDsend",MDsend,1);
}
template <typename T>
MatrixDense<T>::MatrixDense(char ord, size_t m, size_t n, const T *data)
  : MatrixDense<T>(0, ord, m, n, data){}

  // datatype=0: data CPU pointer to _data CPU pointer // NA
  // datatype=1: data GPU pointer to _data GPU pointer
  // datatype=2: data CPU pointer to _data GPU pointer // NA
  // datatype=3: data CPU pointer to _data GPU pointer // NA
template <typename T>
MatrixDense<T>::MatrixDense(int wDev, int datatype, char ord, size_t m, size_t n, T *data)
  : Matrix<T>(m, n, 0), _wDev(wDev), _datatype(datatype),_data(0) {
  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));
  _datay=NULL;
  _vdata=NULL;
  _vdatay=NULL;

  ASSERT(ord == 'r' || ord == 'R' || ord == 'c' || ord == 'C');
  _ord = (ord == 'r' || ord == 'R') ? ROW : COL;

  
  fprintf(stderr,"MatrixDense2: ord=%c m=%d n=%d\n",ord,(int)m,(int)n);
  
#ifdef _DEBUG
  //    CUDACHECK(hipSetDeviceFlags(hipDeviceMapHost)); // TODO: MapHostMemory
  hipDeviceProp_t props;
  CUDACHECK(hipGetDeviceProperties(&props, _wDev));
  fprintf(stderr,"Using: Compute %d.%d CUDA device: [%s] with id=%2d\n", props.major, props.minor, props.name,wDev); fflush(stderr);
#endif
  

  if(datatype==1){
    // no info->orig_data, so send 0 to GpuData
    PUSH_RANGE("MDnew",MDnew,1);
    GpuData<T> *info = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_info = reinterpret_cast<void*>(info);
    POP_RANGE("MDnew",MDnew,1);
    
  // source pointer is on this GPU

    // just copy GPU pointer
    _data = data;
  }
  else{
    PUSH_RANGE("MDnew",MDnew,1);
    GpuData<T> *info = new GpuData<T>(data); // new structure (holds pointer to data and GPU handle)
    this->_info = reinterpret_cast<void*>(info);
    POP_RANGE("MDnew",MDnew,1);

  // Copy CPU Matrix to GPU.
    PUSH_RANGE("MDsend",MDsend,1);
    //  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info); // cast void -> GpuData
    double t0 = timer<double>();
    hipMalloc(&_data, this->_m * this->_n * sizeof(T)); // allocate on GPU
    double t1 = timer<double>();
    hipMemcpy(_data, info->orig_data, this->_m * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
    double t2 = timer<double>();
    printf("Time to allocate the data matrix on the GPU: %f\n", t1-t0);
    printf("Time to copy the data matrix to the GPU    : %f\n", t2-t1);
    POP_RANGE("MDsend",MDsend,1);
  }
}


  // like original MatrixDense, but also feed in CPU data for trainY, validX, and validY
  // Used by elastic_net_mapd.cpp to pass CPU data and put on GPU
template <typename T>
MatrixDense<T>::MatrixDense(int wDev, char ord, size_t m, size_t n, size_t mValid, const T *data, const T *datay, const T *vdata, const T *vdatay)
  : Matrix<T>(m, n, mValid), _wDev(wDev), _datatype(0),_data(0), _datay(0), _vdata(0), _vdatay(0) {
  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));

  ASSERT(ord == 'r' || ord == 'R' || ord == 'c' || ord == 'C');
  _ord = (ord == 'r' || ord == 'R') ? ROW : COL;

  
  fprintf(stderr,"MatrixDense3: ord=%c m=%d n=%d mValid=%d\n",ord,(int)m,(int)n,int(mValid));
  
#ifdef _DEBUG
  //    CUDACHECK(hipSetDeviceFlags(hipDeviceMapHost)); // TODO: MapHostMemory
  hipDeviceProp_t props;
  CUDACHECK(hipGetDeviceProperties(&props, _wDev));
  fprintf(stderr,"Using: Compute %d.%d CUDA device: [%s] with id=%2d\n", props.major, props.minor, props.name,wDev); fflush(stderr);
#endif


  // source pointer is on CPU
  // Set GPU specific _info.
  PUSH_RANGE("MDnew",MDnew,1);
  GpuData<T> *info = new GpuData<T>(data); // new structure (holds pointer to data and GPU handle)
  GpuData<T> *infoy = new GpuData<T>(datay); // new structure (holds pointer to data and GPU handle)
  GpuData<T> *vinfo = new GpuData<T>(vdata); // new structure (holds pointer to data and GPU handle)
  GpuData<T> *vinfoy = new GpuData<T>(vdatay); // new structure (holds pointer to data and GPU handle)
  this->_info = reinterpret_cast<void*>(info);
  this->_infoy = reinterpret_cast<void*>(infoy);
  this->_vinfo = reinterpret_cast<void*>(vinfo);
  this->_vinfoy = reinterpret_cast<void*>(vinfoy);
  POP_RANGE("MDnew",MDnew,1);


  // Copy Matrix to GPU.
  PUSH_RANGE("MDsend",MDsend,1);
  //  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info); // cast void -> GpuData
  double t0 = timer<double>();
  hipMalloc(&_data, this->_m * this->_n * sizeof(T)); // allocate on GPU
  hipMalloc(&_datay, this->_m * sizeof(T)); // allocate on GPU
  hipMalloc(&_vdata, this->_mvalid * this->_n * sizeof(T)); // allocate on GPU
  hipMalloc(&_vdatay, this->_mvalid * sizeof(T)); // allocate on GPU
  double t1 = timer<double>();
  hipMemcpy(_data, info->orig_data, this->_m * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
  hipMemcpy(_datay, infoy->orig_data, this->_m * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
  hipMemcpy(_vdata, vinfo->orig_data, this->_mvalid * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
  hipMemcpy(_vdatay, vinfoy->orig_data, this->_mvalid * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
  double t2 = timer<double>();
  printf("Time to allocate the data matrix on the GPU: %f\n", t1-t0);
  printf("Time to copy the data matrix to the GPU    : %f\n", t2-t1);
  POP_RANGE("MDsend",MDsend,1);
}
  // like original MatrixDense, but also feed in CPU data for trainY, validX, and validY
  // Used by elastic_net_mapd.cpp to pass CPU data and put on GPU
  // datatype=0: CPU pointer to data
  // datatype=1: GPU pointer to data
template <typename T>
MatrixDense<T>::MatrixDense(int wDev, int datatype, char ord, size_t m, size_t n, size_t mValid, T *data, T *datay, T *vdata, T *vdatay)
  : Matrix<T>(m, n, mValid), _wDev(wDev), _datatype(datatype),_data(0), _datay(0), _vdata(0), _vdatay(0) {
  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));

  ASSERT(ord == 'r' || ord == 'R' || ord == 'c' || ord == 'C');
  _ord = (ord == 'r' || ord == 'R') ? ROW : COL;

  
  fprintf(stderr,"ord=%c m=%d n=%d mValid=%d\n",ord,(int)m,(int)n,int(mValid));
  
#ifdef _DEBUG
  //    CUDACHECK(hipSetDeviceFlags(hipDeviceMapHost)); // TODO: MapHostMemory
  hipDeviceProp_t props;
  CUDACHECK(hipGetDeviceProperties(&props, _wDev));
  fprintf(stderr,"Using: Compute %d.%d CUDA device: [%s] with id=%2d\n", props.major, props.minor, props.name,wDev); fflush(stderr);
#endif


  if(datatype==1){
    // source pointer is on GPU already
    // Set GPU specific _info.
    PUSH_RANGE("MDnew",MDnew,1);
    GpuData<T> *info = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *infoy = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *vinfo = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *vinfoy = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_info = reinterpret_cast<void*>(info);
    this->_infoy = reinterpret_cast<void*>(infoy);
    this->_vinfo = reinterpret_cast<void*>(vinfo);
    this->_vinfoy = reinterpret_cast<void*>(vinfoy);
    POP_RANGE("MDnew",MDnew,1);


    // Just copy pointer
    _data = data;
    _datay = datay;
    _vdata = vdata;
    _vdatay = vdatay;
  }
  else{
    // source pointer is on CPU
    // Set GPU specific _info.
    PUSH_RANGE("MDnew",MDnew,1);
    GpuData<T> *info = new GpuData<T>(data); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *infoy = new GpuData<T>(datay); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *vinfo = new GpuData<T>(vdata); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *vinfoy = new GpuData<T>(vdatay); // new structure (holds pointer to data and GPU handle)
    this->_info = reinterpret_cast<void*>(info);
    this->_infoy = reinterpret_cast<void*>(infoy);
    this->_vinfo = reinterpret_cast<void*>(vinfo);
    this->_vinfoy = reinterpret_cast<void*>(vinfoy);
    POP_RANGE("MDnew",MDnew,1);


    // Copy CPU Matrix to GPU.
    PUSH_RANGE("MDsend",MDsend,1);
    //  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info); // cast void -> GpuData
    double t0 = timer<double>();
    hipMalloc(&_data, this->_m * this->_n * sizeof(T)); // allocate on GPU
    hipMalloc(&_datay, this->_m * sizeof(T)); // allocate on GPU
    hipMalloc(&_vdata, this->_mvalid * this->_n * sizeof(T)); // allocate on GPU
    hipMalloc(&_vdatay, this->_mvalid * sizeof(T)); // allocate on GPU
    double t1 = timer<double>();
    hipMemcpy(_data, info->orig_data, this->_m * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
    hipMemcpy(_datay, infoy->orig_data, this->_m * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
    hipMemcpy(_vdata, vinfo->orig_data, this->_mvalid * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
    hipMemcpy(_vdatay, vinfoy->orig_data, this->_mvalid * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
    double t2 = timer<double>();
    printf("Time to allocate the data matrix on the GPU: %f\n", t1-t0);
    printf("Time to copy the data matrix to the GPU    : %f\n", t2-t1);
    POP_RANGE("MDsend",MDsend,1);
  }
}


  // MatrixDense where input actual A object that contains all CPU information, but need to go from 1 GPU to multiple GPU
  // Used by elastic_net_mapd.cpp inside openmp loop for each core
template <typename T>
MatrixDense<T>::MatrixDense(int wDev, const MatrixDense<T>& A)
  : Matrix<T>(A._m, A._n, A._mvalid), _wDev(wDev), _data(0), _ord(A._ord) {

  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));
  
  PUSH_RANGE("MDnew",MDnew,2);
  GpuData<T> *info_A   = reinterpret_cast<GpuData<T>*>(A._info); // cast from void to GpuData
  GpuData<T> *infoy_A  = reinterpret_cast<GpuData<T>*>(A._infoy); // cast from void to GpuData
  GpuData<T> *vinfo_A  = reinterpret_cast<GpuData<T>*>(A._vinfo); // cast from void to GpuData
  GpuData<T> *vinfoy_A = reinterpret_cast<GpuData<T>*>(A._vinfoy); // cast from void to GpuData
  
  GpuData<T> *info;
  GpuData<T> *infoy;
  GpuData<T> *vinfo;
  GpuData<T> *vinfoy;
  if(A._data) info = new GpuData<T>(info_A->orig_data); // create new GpuData structure with point to CPU data
  if(A._datay) infoy  = new GpuData<T>(infoy_A->orig_data); // create new GpuData structure with point to CPU data
  if(A._vdata) vinfo  = new GpuData<T>(vinfo_A->orig_data); // create new GpuData structure with point to CPU data
  if(A._vdatay) vinfoy = new GpuData<T>(vinfoy_A->orig_data); // create new GpuData structure with point to CPU data
  
  this->_info = reinterpret_cast<void*>(info); // back to cast as void
  this->_infoy = reinterpret_cast<void*>(infoy); // back to cast as void
  this->_vinfo = reinterpret_cast<void*>(vinfo); // back to cast as void
  this->_vinfoy = reinterpret_cast<void*>(vinfoy); // back to cast as void
  POP_RANGE("MDnew",MDnew,2);


  if(A._wDev == _wDev){ // if on same device, just copy pointer
    _data   = A._data;
    _datay  = A._datay;
    _vdata  = A._vdata;
    _vdatay = A._vdatay;
  }
  else{
    // Copy Matrix to from source GPU to this GPU
    PUSH_RANGE("MDcopy",MDcopy,1);
    //GpuData<T> *info = reinterpret_cast<GpuData<T>*>(_info); // cast void -> GpuData
    double t0 = timer<double>();
    if(A._data) hipMalloc(&_data, A._m * A._n * sizeof(T)); // allocate on GPU
    if(A._datay) hipMalloc(&_datay, A._m * sizeof(T)); // allocate on GPU
    if(A._vdata) hipMalloc(&_vdata, A._mvalid * A._n * sizeof(T)); // allocate on GPU
    if(A._vdatay) hipMalloc(&_vdatay, A._mvalid * sizeof(T)); // allocate on GPU
    double t1 = timer<double>();
    if(A._data) hipMemcpyPeer(_data, _wDev, A._data, A._wDev, A._m * A._n * sizeof(T)); // dest: _data destid: _wDev  source: A._data sourceid: A._wDev
    if(A._datay) hipMemcpyPeer(_datay, _wDev, A._datay, A._wDev, A._m * sizeof(T)); // dest: _data destid: _wDev  source: A._data sourceid: A._wDev
    if(A._vdata) hipMemcpyPeer(_vdata, _wDev, A._vdata, A._wDev, A._mvalid * A._n * sizeof(T)); // dest: _data destid: _wDev  source: A._data sourceid: A._wDev
    if(A._vdatay) hipMemcpyPeer(_vdatay, _wDev, A._vdatay, A._wDev, A._mvalid * sizeof(T)); // dest: _data destid: _wDev  source: A._data sourceid: A._wDev
    double t2 = timer<double>();
    printf("Time to allocate the data matrix on the GPU: %f\n", t1-t0);
    printf("Time to copy the data matrix to the GPU    : %f\n", t2-t1);
    POP_RANGE("MDcopy",MDcopy,1);
  }
}

template <typename T>
MatrixDense<T>::MatrixDense(const MatrixDense<T>& A)
  : MatrixDense<T>(A._wDev, A){}

template <typename T>
MatrixDense<T>::~MatrixDense() {
  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));
  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);

  delete info;
  this->_info = 0;

  if (this->_done_init && _data) {
    hipFree(_data);
    this->_data = 0;
    DEBUG_CUDA_CHECK_ERR();
  }
  if (this->_done_init && _datay) {
    hipFree(_datay);
    this->_datay = 0;
    DEBUG_CUDA_CHECK_ERR();
  }
  if (this->_done_init && _vdata) {
    hipFree(_vdata);
    this->_vdata = 0;
    DEBUG_CUDA_CHECK_ERR();
  }
  if (this->_done_init && _vdatay) {
    hipFree(_vdatay);
    this->_vdatay = 0;
    DEBUG_CUDA_CHECK_ERR();
  }
}
      
template <typename T>
int MatrixDense<T>::Init() {
  DEBUG_EXPECT(!this->_done_init);
  if (this->_done_init)
    return 1;
  this->_done_init = true;
  CUDACHECK(hipSetDevice(_wDev));

  PUSH_RANGE("MDinit",MDinit,1);
  POP_RANGE("MDinit",MDinit,1);

  DEBUG_CUDA_CHECK_ERR();

  return 0;
}

template <typename T>
void MatrixDense<T>::GetTrainX(int datatype, size_t size, T**data) const {

  CUDACHECK(hipSetDevice(_wDev));

  if(datatype==1){
    hipMemcpy(*data, _data, size* sizeof(T),hipMemcpyDeviceToHost);
    CUDA_CHECK_ERR();
  }
  else{
    std::memcpy(*data, _data, size * sizeof(T));
  }

  return;
}
template <typename T>
void MatrixDense<T>::GetTrainY(int datatype, size_t size, T**data) const {

  CUDACHECK(hipSetDevice(_wDev));

  if(datatype==1){
    hipMemcpy(*data, _datay, size* sizeof(T),hipMemcpyDeviceToHost);
    CUDA_CHECK_ERR();
  }
  else{
    std::memcpy(*data, _datay, size * sizeof(T));
  }

  return;
}

template <typename T>
void MatrixDense<T>::GetValidX(int datatype, size_t size, T**data) const {

  CUDACHECK(hipSetDevice(_wDev));

  if(datatype==1){
    hipMemcpy(*data, _vdata, size* sizeof(T),hipMemcpyDeviceToHost);
    CUDA_CHECK_ERR();
  }
  else{
    std::memcpy(*data, _vdata, size * sizeof(T));
  }

  return;
}
template <typename T>
void MatrixDense<T>::GetValidY(int datatype, size_t size, T**data) const {

  CUDACHECK(hipSetDevice(_wDev));

  if(datatype==1){
    hipMemcpy(*data, _vdatay, size* sizeof(T),hipMemcpyDeviceToHost);
    CUDA_CHECK_ERR();
  }
  else{
    std::memcpy(*data, _vdatay, size * sizeof(T));
  }


  return;
}


template <typename T>
int MatrixDense<T>::Mul(char trans, T alpha, const T *x, T beta, T *y) const {

  DEBUG_EXPECT(this->_done_init);
  if (!this->_done_init)
    return 1;
  CUDACHECK(hipSetDevice(_wDev));

  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  hipblasHandle_t hdl = info->handle;

  const cml::vector<T> x_vec = cml::vector_view_array<T>(x, this->_n);
  cml::vector<T> y_vec = cml::vector_view_array<T>(y, this->_m);

  //  Performs the matrix-vector operations y := alpha*A*x + beta*y or y := alpha*A'*x + beta*y where alpha and beta are scalars, x and y are vectors and A is an m by n matrix
  //https://docs.oracle.com/cd/B19306_01/appdev.102/b14258/u_nla.htm#CIAFEAFG
  if (_ord == ROW) {
    cml::matrix<T, CblasRowMajor> A =
        cml::matrix_view_array<T, CblasRowMajor>(_data, this->_m, this->_n);
    cml::blas_gemv(hdl, OpToCublasOp(trans), alpha, &A, &x_vec, beta,
        &y_vec);
  } else {
    cml::matrix<T, CblasColMajor> A =
        cml::matrix_view_array<T, CblasColMajor>(_data, this->_m, this->_n);
    cml::blas_gemv(hdl, OpToCublasOp(trans), alpha, &A, &x_vec, beta, &y_vec);
  }
  CUDA_CHECK_ERR();

  return 0;
}


  // Equilibration (precondition) matrix using Sinkhorn Knopp method wrapped to allow any norm
  // See https://arxiv.org/pdf/1610.03871.pdf for more information
template <typename T>
int MatrixDense<T>::Equil(T *d, T *e, bool equillocal) {
  DEBUG_ASSERT(this->_done_init);
  if (!this->_done_init)
    return 1;

  CUDACHECK(hipSetDevice(_wDev));

  // Extract cublas handle from _info.
  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  hipblasHandle_t hdl = info->handle;

  // Number of elements in matrix.
  size_t num_el = this->_m * this->_n;

  // Create bit-vector with signs of entries in A and then let A = f(A),
  // where f = |A| or f = |A|.^2.
  unsigned char *sign;
  size_t num_sign_bytes = (num_el + 7) / 8;
  hipMalloc(&sign, num_sign_bytes);
  CUDA_CHECK_ERR();

  // Fill sign bits, assigning each thread a multiple of 8 elements.
  size_t num_chars = num_el / 8;
  size_t grid_size = cml::calc_grid_dim(num_chars, cml::kBlockSize);
  if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
    __SetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
        SquareF<T>());
  } else {
    __SetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
        AbsF<T>());
  }
  wrapcudaDeviceSynchronize();
  CUDA_CHECK_ERR();

  // If numel(A) is not a multiple of 8, then we need to set the last couple
  // of sign bits too. 
  if (num_el > num_chars * 8) {
    if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
      __SetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
          num_el - num_chars * 8, SquareF<T>());
    } else {
      __SetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
          num_el - num_chars * 8, AbsF<T>());
    }
    wrapcudaDeviceSynchronize();
    CUDA_CHECK_ERR();
  }

  // Perform Sinkhorn-Knopp equilibration to obtain a doubly stochastic matrix.
  SinkhornKnopp(this, d, e, equillocal);
  wrapcudaDeviceSynchronize();

  // Transform A = sign(A) .* sqrt(A) if 2-norm equilibration was performed,
  // or A = sign(A) .* A if the 1-norm was equilibrated.
  if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
    __UnSetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
        SqrtF<T>());
  } else {
    __UnSetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
        IdentityF<T>());
  }
  wrapcudaDeviceSynchronize();
  CUDA_CHECK_ERR();

  // Deal with last few entries if num_el is not a multiple of 8.
  if (num_el > num_chars * 8) {
    if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
      __UnSetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
          num_el - num_chars * 8, SqrtF<T>());
    } else {
      __UnSetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
          num_el - num_chars * 8, IdentityF<T>());
    }
    wrapcudaDeviceSynchronize();
    CUDA_CHECK_ERR();
  }

  // Compute D := sqrt(D), E := sqrt(E), if 2-norm was equilibrated.
  if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
    thrust::transform(thrust::device_pointer_cast(d),
        thrust::device_pointer_cast(d + this->_m),
        thrust::device_pointer_cast(d), SqrtF<T>());
    thrust::transform(thrust::device_pointer_cast(e),
        thrust::device_pointer_cast(e + this->_n),
        thrust::device_pointer_cast(e), SqrtF<T>());
    wrapcudaDeviceSynchronize();
    CUDA_CHECK_ERR();
  }

  // Compute A := D * A * E.
  MultDiag(d, e, this->_m, this->_n, _ord, _data);
  wrapcudaDeviceSynchronize();
  CUDA_CHECK_ERR();

  // Scale A to have norm of 1 (in the kNormNormalize norm).
  T normA = NormEst(hdl, kNormNormalize, *this);
  CUDA_CHECK_ERR();
  wrapcudaDeviceSynchronize();
  cml::vector<T> a_vec = cml::vector_view_array(_data, num_el);
  cml::vector_scale(&a_vec, 1 / normA);
  wrapcudaDeviceSynchronize();

  // Scale d and e to account for normalization of A.
  cml::vector<T> d_vec = cml::vector_view_array<T>(d, this->_m);
  cml::vector<T> e_vec = cml::vector_view_array<T>(e, this->_n);
  cml::vector_scale(&d_vec, 1 / sqrt(normA));
  cml::vector_scale(&e_vec, 1 / sqrt(normA));
  wrapcudaDeviceSynchronize();

  DEBUG_PRINTF("norm A = %e, normd = %e, norme = %e\n", normA,
      cml::blas_nrm2(hdl, &d_vec), cml::blas_nrm2(hdl, &e_vec));

  hipFree(sign);
  CUDA_CHECK_ERR();

  return 0;
}

////////////////////////////////////////////////////////////////////////////////
/////////////////////// Equilibration Helpers //////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
namespace {

// Estimates norm of A. norm_type should either be kNorm2 or kNormFro.
template <typename T>
T NormEst(hipblasHandle_t hdl, NormTypes norm_type, const MatrixDense<T>& A) {
  switch (norm_type) {
    case kNorm2: {
      return Norm2Est(hdl, &A);
    }
    case kNormFro: {
      const cml::vector<T> a = cml::vector_view_array(A.Data(),
          A.Rows() * A.Cols());
      return cml::blas_nrm2(hdl, &a) / std::sqrt(std::min(A.Rows(), A.Cols()));
    }
    case kNorm1:
      // 1-norm normalization doens't make make sense since it treats rows and
      // columns differently.
    default:
      ASSERT(false);
      return static_cast<T>(0.);
  }
}

// Performs A := D * A * E for A in row major
template <typename T>
void __global__ __MultRow(size_t m, size_t n, const T *d, const T *e, T *data) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t t = tid; t < m * n; t += gridDim.x * blockDim.x)
    data[t] *= d[t / n] * e[t % n];
}

// Performs A := D * A * E for A in col major
template <typename T>
void __global__ __MultCol(size_t m, size_t n, const T *d, const T *e, T *data) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t t = tid; t < m * n; t += gridDim.x * blockDim.x)
    data[t] *= d[t % m] * e[t / m];
}

template <typename T>
void MultDiag(const T *d, const T *e, size_t m, size_t n,
              typename MatrixDense<T>::Ord ord, T *data) {
  if (ord == MatrixDense<T>::ROW) {
    size_t grid_dim_row = cml::calc_grid_dim(m * n, cml::kBlockSize);
    __MultRow<<<grid_dim_row, cml::kBlockSize>>>(m, n, d, e, data);
  } else {
    size_t grid_dim_row = cml::calc_grid_dim(m * n, cml::kBlockSize);
    __MultCol<<<grid_dim_row, cml::kBlockSize>>>(m, n, d, e, data);
  }
}

}  // namespace

// Explicit template instantiation.
#if !defined(POGS_DOUBLE) || POGS_DOUBLE==1
template class MatrixDense<double>;
#endif

#if !defined(POGS_SINGLE) || POGS_SINGLE==1
template class MatrixDense<float>;
#endif

}  // namespace pogs

