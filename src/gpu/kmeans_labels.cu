#include "hip/hip_runtime.h"
// original code from https://github.com/NVIDIA/kmeans (Apache V2.0 License)
#include <unistd.h>
#include "kmeans_labels.h"
#include <thrust/device_vector.h>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <sstream>
#include <hipblas.h>
#include <cfloat>
#include "include/kmeans_general.h"

#ifdef CUBLAS_API_H_
// cuBLAS API errors
static const char *cudaGetErrorEnum(hipblasStatus_t error) {
    switch (error) {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
#endif

extern hipStream_t hip_stream[MAX_NGPUS];

template<unsigned int i>
extern __global__ void debugMark() {};

hipStream_t hip_stream[MAX_NGPUS];
namespace kmeans {
    namespace detail {

        template<typename T>
        struct absolute_value {
            __host__ __device__

            void operator()(T &x) const {
                x = (x > 0 ? x : -x);
            }
        };


        hipblasHandle_t cublas_handle[MAX_NGPUS];

        void labels_init() {
            hipblasStatus_t stat;
            hipError_t err;
            int dev_num;
            safe_cuda(hipGetDevice(&dev_num));
            stat = hipblasCreate(&detail::cublas_handle[dev_num]);
            if (stat != HIPBLAS_STATUS_SUCCESS) {
                std::cout << "CUBLAS initialization failed" << std::endl;
                exit(1);
            }
            err = safe_cuda(hipStreamCreate(&hip_stream[dev_num]));
            if (err != hipSuccess) {
                std::cout << "Stream creation failed" << std::endl;

            }
            hipblasSetStream(cublas_handle[dev_num], hip_stream[dev_num]);
            mycub::cub_init();
        }

        void labels_close() {
            int dev_num;
            safe_cuda(hipGetDevice(&dev_num));
            safe_cublas(hipblasDestroy(cublas_handle[dev_num]));
            safe_cuda(hipStreamDestroy(hip_stream[dev_num]));
            mycub::cub_close();
        }

        template<typename T>
        void memcpy(thrust::host_vector <T, std::allocator<T>> &H,
                    thrust::device_vector <T, thrust::device_malloc_allocator<T>> &D) {
            int dev_num;
            safe_cuda(hipGetDevice(&dev_num));
            safe_cuda(hipMemcpyAsync(thrust::raw_pointer_cast(H.data()),
                                      thrust::raw_pointer_cast(D.data()),
                                      sizeof(T) * D.size(), hipMemcpyDeviceToHost, hip_stream[dev_num]));
        }

        template<typename T>
        void memcpy(thrust::device_vector <T, thrust::device_malloc_allocator<T>> &D,
                    thrust::host_vector <T, std::allocator<T>> &H) {
            int dev_num;
            safe_cuda(hipGetDevice(&dev_num));
            safe_cuda(hipMemcpyAsync(thrust::raw_pointer_cast(D.data()),
                                      thrust::raw_pointer_cast(H.data()),
                                      sizeof(T) * H.size(), hipMemcpyHostToDevice, hip_stream[dev_num]));
        }

        template<typename T>
        void memcpy(thrust::device_vector <T, thrust::device_malloc_allocator<T>> &Do,
                    thrust::device_vector <T, thrust::device_malloc_allocator<T>> &Di) {
            int dev_num;
            safe_cuda(hipGetDevice(&dev_num));
            safe_cuda(hipMemcpyAsync(thrust::raw_pointer_cast(Do.data()),
                                      thrust::raw_pointer_cast(Di.data()),
                                      sizeof(T) * Di.size(), hipMemcpyDeviceToDevice, hip_stream[dev_num]));
        }

        template<typename T>
        void memzero(thrust::device_vector <T, thrust::device_malloc_allocator<T>> &D) {
            int dev_num;
            safe_cuda(hipGetDevice(&dev_num));
            safe_cuda(
                    hipMemsetAsync(thrust::raw_pointer_cast(D.data()), 0, sizeof(T) * D.size(), hip_stream[dev_num]));
        }

        void streamsync(int dev_num) {
            hipStreamSynchronize(hip_stream[dev_num]);
        }

        template<typename T>
        __global__ void self_dots(int n, int d, T *data, T *dots) {
            T accumulator = 0;
            int global_id = blockDim.x * blockIdx.x + threadIdx.x;

            if (global_id < n) {
                for (int i = 0; i < d; i++) {
                    T value = data[i + global_id * d];
                    accumulator += value * value;
                }
                dots[global_id] = accumulator;
            }
        }

        template<typename T>
        void make_self_dots(int n, int d, thrust::device_vector <T> &data, thrust::device_vector <T> &dots) {
            int dev_num;
#define MAX_BLOCK_THREADS0 256
            const int GRID_SIZE = (n - 1) / MAX_BLOCK_THREADS0 + 1;
            safe_cuda(hipGetDevice(&dev_num));
            self_dots << < GRID_SIZE, MAX_BLOCK_THREADS0, 0, hip_stream[dev_num] >> >
                                                             (n, d, thrust::raw_pointer_cast(data.data()),
                                                                     thrust::raw_pointer_cast(dots.data()));
#if(CHECK)
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
#endif

        }

#define MAX_BLOCK_THREADS 32

        template<typename T>
        __global__ void all_dots(int n, int k, T *data_dots, T *centroid_dots, T *dots) {
            __shared__
            T local_data_dots[MAX_BLOCK_THREADS];
            __shared__
            T local_centroid_dots[MAX_BLOCK_THREADS];
            //        if(threadIdx.x==0 && threadIdx.y==0 && blockIdx.x==0) printf("inside %d %d %d\n",threadIdx.x,blockIdx.x,blockDim.x);

            int data_index = threadIdx.x + blockIdx.x * blockDim.x;
            if ((data_index < n) && (threadIdx.y == 0)) {
                local_data_dots[threadIdx.x] = data_dots[data_index];
            }

            int centroid_index = threadIdx.x + blockIdx.y * blockDim.y;
            if ((centroid_index < k) && (threadIdx.y == 1)) {
                local_centroid_dots[threadIdx.x] = centroid_dots[centroid_index];
            }

            __syncthreads();

            centroid_index = threadIdx.y + blockIdx.y * blockDim.y;
            //        printf("data_index=%d centroid_index=%d\n",data_index,centroid_index);
            if ((data_index < n) && (centroid_index < k)) {
                dots[data_index + centroid_index * n] = local_data_dots[threadIdx.x] +
                                                        local_centroid_dots[threadIdx.y];
            }
        }


        template<typename T>
        void make_all_dots(int n, int k, thrust::device_vector <T> &data_dots,
                           thrust::device_vector <T> &centroid_dots,
                           thrust::device_vector <T> &dots) {
            int dev_num;
            safe_cuda(hipGetDevice(&dev_num));
            const int BLOCK_THREADSX = MAX_BLOCK_THREADS; // BLOCK_THREADSX*BLOCK_THREADSY<=1024 on modern arch's (sm_61)
            const int BLOCK_THREADSY = MAX_BLOCK_THREADS;
            const int GRID_SIZEX = (n - 1) / BLOCK_THREADSX + 1; // on old arch's this has to be less than 2^16=65536
            const int GRID_SIZEY = (k - 1) / BLOCK_THREADSY + 1; // this has to be less than 2^16=65536
            //        printf("pre all_dots: %d %d %d %d\n",GRID_SIZEX,GRID_SIZEY,BLOCK_THREADSX,BLOCK_THREADSY); fflush(stdout);
            all_dots << <
            dim3(GRID_SIZEX, GRID_SIZEY),
                    dim3(BLOCK_THREADSX, BLOCK_THREADSY), 0,
                    hip_stream[dev_num] >> > (n, k, thrust::raw_pointer_cast(data_dots.data()),
                            thrust::raw_pointer_cast(centroid_dots.data()),
                            thrust::raw_pointer_cast(dots.data()));
#if(CHECK)
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
#endif
        };

        template<>
        void calculate_distances<double>(int verbose, int q, int n, int d, int k,
                                         thrust::device_vector<double> &data,
                                         thrust::device_vector<double> &centroids,
                                         thrust::device_vector<double> &data_dots,
                                         thrust::device_vector<double> &centroid_dots,
                                         thrust::device_vector<double> &pairwise_distances) {
            detail::make_self_dots(k, d, centroids, centroid_dots);
            detail::make_all_dots(n, k, data_dots, centroid_dots, pairwise_distances);
            //||x-y||^2 = ||x||^2 + ||y||^2 - 2 x . y
            //pairwise_distances has ||x||^2 + ||y||^2, so beta = 1
            //The dgemm calculates x.y for all x and y, so alpha = -2.0
            double alpha = -2.0;
            double beta = 1.0;
            //If the data were in standard column major order, we'd do a
            //centroids * data ^ T
            //But the data is in row major order, so we have to permute
            //the arguments a little
            int dev_num;
            safe_cuda(hipGetDevice(&dev_num));
            hipblasStatus_t stat =
            safe_cublas(hipblasDgemm(detail::cublas_handle[dev_num],
                                    HIPBLAS_OP_T, HIPBLAS_OP_N,
                                    n, k, d, &alpha,
                                    thrust::raw_pointer_cast(data.data()),
                                    d,//Has to be n or d
                                    thrust::raw_pointer_cast(centroids.data()),
                                    d,//Has to be k or d
                                    &beta,
                                    thrust::raw_pointer_cast(pairwise_distances.data()),
                                    n)); //Has to be n or k

            thrust::for_each(pairwise_distances.begin(), pairwise_distances.end(),
                             absolute_value<double>()); // in-place transformation to ensure all distances are positive indefinite

            if (stat != HIPBLAS_STATUS_SUCCESS) {
                std::cout << "Invalid Dgemm" << std::endl;
                exit(1);
            }

        }

        template<>
        void calculate_distances<float>(int verbose, int q, int n, int d, int k,
                                        thrust::device_vector<float> &data,
                                        thrust::device_vector<float> &centroids,
                                        thrust::device_vector<float> &data_dots,
                                        thrust::device_vector<float> &centroid_dots,
                                        thrust::device_vector<float> &pairwise_distances) {
            detail::make_self_dots(k, d, centroids, centroid_dots);
            detail::make_all_dots(n, k, data_dots, centroid_dots, pairwise_distances);

            if (verbose) {
                thrust::host_vector<float> h_data_dots = data_dots;
                thrust::host_vector<float> h_centroid_dots = centroid_dots;
                thrust::host_vector<float> h_pairwise_distances = pairwise_distances;

                for (int i = 0; i < n; i++) {
                    if (i % 1 == 0) {
                        fprintf(stderr, "0 q=%d data_dots[%d]=%g\n", q, i, h_data_dots[i]);
                        fflush(stderr);
                    }
                }
                for (int i = 0; i < k; i++) {
                    fprintf(stderr, "0 q=%d centroid_dots[%d]=%g\n", q, i, h_centroid_dots[i]);
                    fflush(stderr);
                }
                for (int i = 0; i < n * k; i++) {
                    if (i % 1 == 0) {
                        fprintf(stderr, "0 q=%d pairwise_distances[%d]=%g\n", q, i, h_pairwise_distances[i]);
                        fflush(stderr);
                    }
                }
            }

            //||x-y||^2 = ||x||^2 + ||y||^2 - 2 x . y
            //pairwise_distances has ||x||^2 + ||y||^2, so beta = 1
            //The dgemm calculates x.y for all x and y, so alpha = -2.0
            float alpha = -2.0;
            float beta = 1.0;
            //If the data were in standard column major order, we'd do a
            //centroids * data ^ T
            //But the data is in row major order, so we have to permute
            //the arguments a little
            int dev_num;
            safe_cuda(hipGetDevice(&dev_num));
            // http://docs.nvidia.com/cuda/cublas/index.html#axzz4kgBuzSr6
            hipblasStatus_t stat;
            if (0) {
                int M = n;
                int N = k;
                int K = d;
                int lda = K;
                int ldb = N;
                int ldc = M;
                fprintf(stderr, "%d x %d : data size=%zu\n", lda, M, data.size());
                fflush(stderr);
                fprintf(stderr, "%d x %d : centroids size=%zu\n", ldb, K, centroids.size());
                fflush(stderr);
                fprintf(stderr, "%d x %d : pairwise_distances size=%zu\n", ldc, N, pairwise_distances.size());
                fflush(stderr);
                stat =
                safe_cublas(hipblasSgemm(detail::cublas_handle[dev_num],
                                        HIPBLAS_OP_T, HIPBLAS_OP_T,
                                        M, N, K, &alpha,
                                        thrust::raw_pointer_cast(
                                                data.data()), // <type> array of dimensions lda x k with lda>=max(1,m) if transa == HIPBLAS_OP_N and lda x m with lda>=max(1,k) otherwise.
                                        lda,
                                        thrust::raw_pointer_cast(
                                                centroids.data()), // <type> array of dimension ldb x n with ldb>=max(1,k) if transa == HIPBLAS_OP_N and ldb x k with ldb>=max(1,n) otherwise.
                                        ldb,
                                        &beta,
                                        thrust::raw_pointer_cast(
                                                pairwise_distances.data()), // <type> array of dimensions ldc x n with ldc>=max(1,m).
                                        ldc));
            } else if (0) {
                int M = n;
                int N = k;
                int K = d;
                int lda = M;
                int ldb = K;
                int ldc = M;
                fprintf(stderr, "A2 %d x %d : data size=%zu\n", lda, K, data.size());
                fflush(stderr);
                fprintf(stderr, "B2 %d x %d : centroids size=%zu\n", ldb, N, centroids.size());
                fflush(stderr);
                fprintf(stderr, "C2 %d x %d : pairwise_distances size=%zu\n", ldc, N, pairwise_distances.size());
                fflush(stderr);
                stat =
                safe_cublas(hipblasSgemm(detail::cublas_handle[dev_num],
                                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                                        M, N, K, &alpha,
                                        thrust::raw_pointer_cast(
                                                data.data()), // <type> array of dimensions lda x k with lda>=max(1,m) if transa == HIPBLAS_OP_N and lda x m with lda>=max(1,k) otherwise.
                                        lda,
                                        thrust::raw_pointer_cast(
                                                centroids.data()), // <type> array of dimension ldb x n with ldb>=max(1,k) if transa == HIPBLAS_OP_N and ldb x k with ldb>=max(1,n) otherwise.
                                        ldb,
                                        &beta,
                                        thrust::raw_pointer_cast(
                                                pairwise_distances.data()), // <type> array of dimensions ldc x n with ldc>=max(1,m).
                                        ldc));
            } else {
                int M = n; // rows in op(A) and C
                int N = k; // cols in op(B) and C
                int K = d; // cols in op(A) and op(B)
                int lda = K;
                int ldb = K; // http://docs.nvidia.com/cuda/cublas/index.html#axzz4kgBuzSr6 has mistake, transa should have been transb
                //see http://www.netlib.org/lapack/explore-html/db/dc9/group__single__blas__level3_gafe51bacb54592ff5de056acabd83c260.html#gafe51bacb54592ff5de056acabd83c260
                int ldc = M;
                if (verbose >= 2) {
                    fprintf(stderr, "A3 %d x %d -> %d x %d : data size=%zu\n", K, M, M, K, data.size());
                    fflush(stderr);
                    fprintf(stderr, "B3 %d x %d -> %d x %d : centroids size=%zu\n", K, N, K, N, centroids.size());
                    fflush(stderr);
                    fprintf(stderr, "C3 %d x %d : pairwise_distances size=%zu\n", M, N, pairwise_distances.size());
                    fflush(stderr);
                    fflush(stderr);
                    //sleep(5);
                }
                stat =
                safe_cublas(hipblasSgemm(detail::cublas_handle[dev_num],
                                        HIPBLAS_OP_T, HIPBLAS_OP_N,
                                        M, N, K, &alpha,
                                        thrust::raw_pointer_cast(data.data()),
                                        lda,//Has to be n or d
                                        thrust::raw_pointer_cast(centroids.data()),
                                        ldb,//Has to be k or d
                                        &beta,
                                        thrust::raw_pointer_cast(pairwise_distances.data()),
                                        ldc)); //Has to be n or k
                if (verbose >= 2) {
                    fprintf(stderr, "After hipblasSgemm\n");
                    fflush(stderr);
                    //sleep(5);
                }

                thrust::for_each(pairwise_distances.begin(), pairwise_distances.end(),
                                 absolute_value<float>()); // in-place transformation to ensure all distances are positive indefinite
                if (verbose) {
                    thrust::host_vector<float> h_data = data;
                    thrust::host_vector<float> h_centroids = centroids;
                    thrust::host_vector<float> h_pairwise_distances = pairwise_distances;

                    for (int i = 0; i < M * K; i++) {
                        if (i % 1 == 0) {
                            fprintf(stderr, "q=%d data[%d]=%g\n", q, i, h_data[i]);
                            fflush(stderr);
                        }
                    }
                    for (int i = 0; i < K * N; i++) {
                        fprintf(stderr, "q=%d centroids[%d]=%g\n", q, i, h_centroids[i]);
                    }
                    for (int i = 0; i < M * N; i++) {
                        if (i % 1 == 0) {
                            fprintf(stderr, "q=%d pairwise_distances[%d]=%g\n", q, i, h_pairwise_distances[i]);
                            fflush(stderr);
                        }
                    }
                }

            }
            if (stat != HIPBLAS_STATUS_SUCCESS) {
                std::cout << "Invalid Sgemm" << std::endl;
                exit(1);
            }
        }

        template<typename T>
        __global__ void make_new_labels(int n, int k, T *pairwise_distances,
                                        int *labels, int *changes,
                                        T *distances) {
            T min_distance = FLT_MAX; //std::numeric_limits<T>::max(); // might be ok TODO FIXME
            T min_idx = -1;
            int global_id = threadIdx.x + blockIdx.x * blockDim.x;
            if (global_id < n) {
                int old_label = labels[global_id];
                for (int c = 0; c < k; c++) {
                    T distance = pairwise_distances[c * n + global_id];
                    if (distance < min_distance) {
                        min_distance = distance;
                        min_idx = c;
                    }
                }
                labels[global_id] = min_idx;
                distances[global_id] = min_distance;
                if (old_label != min_idx) {
                    atomicAdd(changes, 1);
                }
            }
        }

        template<typename T>
        void relabel(int n, int k,
                     thrust::device_vector <T> &pairwise_distances,
                     thrust::device_vector<int> &labels,
                     thrust::device_vector <T> &distances,
                     int *d_changes) {
            int dev_num;
            safe_cuda(hipGetDevice(&dev_num));
            safe_cuda(hipMemsetAsync(d_changes, 0, sizeof(int), hip_stream[dev_num]));
#define MAX_BLOCK_THREADS2 256
            const int GRID_SIZE = (n - 1) / MAX_BLOCK_THREADS2 + 1;
            make_new_labels << < GRID_SIZE, MAX_BLOCK_THREADS2, 0, hip_stream[dev_num] >> > (
                    n, k,
                            thrust::raw_pointer_cast(pairwise_distances.data()),
                            thrust::raw_pointer_cast(labels.data()),
                            d_changes,
                            thrust::raw_pointer_cast(distances.data()));
#if(CHECK)
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
#endif
        }

    }
}

namespace mycub {
    void *d_key_alt_buf[MAX_NGPUS];
    unsigned int key_alt_buf_bytes[MAX_NGPUS];
    void *d_value_alt_buf[MAX_NGPUS];
    unsigned int value_alt_buf_bytes[MAX_NGPUS];
    void *d_temp_storage[MAX_NGPUS];
    size_t temp_storage_bytes[MAX_NGPUS];
    void *d_temp_storage2[MAX_NGPUS];
    size_t temp_storage_bytes2[MAX_NGPUS];
    bool cub_initted;

    void cub_init() {
        // std::cout <<"CUB init" << std::endl;
        for (int q = 0; q < MAX_NGPUS; q++) {
            d_key_alt_buf[q] = NULL;
            key_alt_buf_bytes[q] = 0;
            d_value_alt_buf[q] = NULL;
            value_alt_buf_bytes[q] = 0;
            d_temp_storage[q] = NULL;
            temp_storage_bytes[q] = 0;
            d_temp_storage2[q] = NULL;
            temp_storage_bytes2[q] = 0;
        }
        cub_initted = true;
    }

    void cub_close() {
        for (int q = 0; q < MAX_NGPUS; q++) {
            if (d_key_alt_buf[q]) safe_cuda(hipFree(d_key_alt_buf[q]));
            if (d_value_alt_buf[q]) safe_cuda(hipFree(d_value_alt_buf[q]));
            if (d_temp_storage[q]) safe_cuda(hipFree(d_temp_storage[q]));
            if (d_temp_storage2[q]) safe_cuda(hipFree(d_temp_storage2[q]));
            d_temp_storage[q] = NULL;
            d_temp_storage2[q] = NULL;
        }
        cub_initted = false;
    }

    void sort_by_key_int(thrust::device_vector<int> &keys, thrust::device_vector<int> &values) {
        int dev_num;
        safe_cuda(hipGetDevice(&dev_num));
        hipStream_t this_stream = hip_stream[dev_num];
        int SIZE = keys.size();
        //int *d_key_alt_buf, *d_value_alt_buf;
        if (key_alt_buf_bytes[dev_num] < sizeof(int) * SIZE) {
            if (d_key_alt_buf[dev_num]) safe_cuda(hipFree(d_key_alt_buf[dev_num]));
            safe_cuda(hipMalloc(&d_key_alt_buf[dev_num], sizeof(int) * SIZE));
            key_alt_buf_bytes[dev_num] = sizeof(int) * SIZE;
        }
        if (value_alt_buf_bytes[dev_num] < sizeof(int) * SIZE) {
            if (d_value_alt_buf[dev_num]) safe_cuda(hipFree(d_value_alt_buf[dev_num]));
            safe_cuda(hipMalloc(&d_value_alt_buf[dev_num], sizeof(int) * SIZE));
            value_alt_buf_bytes[dev_num] = sizeof(int) * SIZE;
        }
        hipcub::DoubleBuffer<int> d_keys(thrust::raw_pointer_cast(keys.data()), (int *) d_key_alt_buf[dev_num]);
        hipcub::DoubleBuffer<int> d_values(thrust::raw_pointer_cast(values.data()), (int *) d_value_alt_buf[dev_num]);

        // Determine temporary device storage requirements for sorting operation
        if (!d_temp_storage[dev_num]) {
            hipcub::DeviceRadixSort::SortPairs(d_temp_storage[dev_num], temp_storage_bytes[dev_num], d_keys,
                                            d_values, SIZE, 0, sizeof(int) * 8, this_stream);
            // Allocate temporary storage for sorting operation
            safe_cuda(hipMalloc(&d_temp_storage[dev_num], temp_storage_bytes[dev_num]));
        }
        // Run sorting operation
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage[dev_num], temp_storage_bytes[dev_num], d_keys,
                                        d_values, SIZE, 0, sizeof(int) * 8, this_stream);
        // Sorted keys and values are referenced by d_keys.Current() and d_values.Current()
    }

    template<typename T, typename U>
    void sort_by_key(thrust::device_vector <T> &keys, thrust::device_vector <U> &values) {
        int dev_num;
        safe_cuda(hipGetDevice(&dev_num));
        hipStream_t this_stream = hip_stream[dev_num];
        int SIZE = keys.size();
        if (key_alt_buf_bytes[dev_num] < sizeof(T) * SIZE) {
            if (d_key_alt_buf[dev_num]) safe_cuda(hipFree(d_key_alt_buf[dev_num]));
            safe_cuda(hipMalloc(&d_key_alt_buf[dev_num], sizeof(T) * SIZE));
            key_alt_buf_bytes[dev_num] = sizeof(T) * SIZE;
            std::cout << "Malloc key_alt_buf" << std::endl;
        }
        if (value_alt_buf_bytes[dev_num] < sizeof(U) * SIZE) {
            if (d_value_alt_buf[dev_num]) safe_cuda(hipFree(d_value_alt_buf[dev_num]));
            safe_cuda(hipMalloc(&d_value_alt_buf[dev_num], sizeof(U) * SIZE));
            value_alt_buf_bytes[dev_num] = sizeof(U) * SIZE;
            std::cout << "Malloc value_alt_buf" << std::endl;
        }
        hipcub::DoubleBuffer <T> d_keys(thrust::raw_pointer_cast(keys.data()), (T *) d_key_alt_buf[dev_num]);
        hipcub::DoubleBuffer <U> d_values(thrust::raw_pointer_cast(values.data()), (U *) d_value_alt_buf[dev_num]);
        hipError_t err;

        // Determine temporary device storage requirements for sorting operation
        //if (temp_storage_bytes[dev_num] == 0) {
        void *d_temp;
        size_t temp_bytes;
        err = hipcub::DeviceRadixSort::SortPairs(d_temp_storage[dev_num], temp_bytes, d_keys,
                                              d_values, SIZE, 0, sizeof(T) * 8, this_stream);
        // Allocate temporary storage for sorting operation
        safe_cuda(hipMalloc(&d_temp, temp_bytes));
        d_temp_storage[dev_num] = d_temp;
        temp_storage_bytes[dev_num] = temp_bytes;
        std::cout << "Malloc temp_storage. " << temp_storage_bytes[dev_num] << " bytes" << std::endl;
        std::cout << "d_temp_storage[" << dev_num << "] = " << d_temp_storage[dev_num] << std::endl;
        if (err) {
            std::cout << "Error " << err << " in SortPairs 1" << std::endl;
            std::cout << hipGetErrorString(err) << std::endl;
        }
        //}
        // Run sorting operation
        err = hipcub::DeviceRadixSort::SortPairs(d_temp, temp_bytes, d_keys,
                                              d_values, SIZE, 0, sizeof(T) * 8, this_stream);
        if (err) std::cout << "Error in SortPairs 2" << std::endl;
        //hipcub::DeviceRadixSort::SortPairs(d_temp_storage[dev_num], temp_storage_bytes[dev_num], d_keys,
        //                                d_values, SIZE, 0, sizeof(T)*8, this_stream);

    }

    template<typename T>
    void sum_reduce(thrust::device_vector <T> &values, T *sum) {
        int dev_num;
        safe_cuda(hipGetDevice(&dev_num));
        if (!d_temp_storage2[dev_num]) {
            hipcub::DeviceReduce::Sum(d_temp_storage2[dev_num], temp_storage_bytes2[dev_num],
                                   thrust::raw_pointer_cast(values.data()),
                                   sum, values.size(), hip_stream[dev_num]);
            // Allocate temporary storage for sorting operation
            safe_cuda(hipMalloc(&d_temp_storage2[dev_num], temp_storage_bytes2[dev_num]));
        }
        hipcub::DeviceReduce::Sum(d_temp_storage2[dev_num], temp_storage_bytes2[dev_num],
                               thrust::raw_pointer_cast(values.data()),
                               sum, values.size(), hip_stream[dev_num]);
    }
}
